//
// Wrapper for cublasDgemm function. 
//
// Alan Gray, NVIDIA
//


#include <hip/hip_runtime.h>
#include <stdio.h>
#include "hipblas.h" 


bool alreadyAllocated_dgemm=false;
bool alreadyAllocated_dgemm_handle=false;

double **d_Aarray;
double **d_Barray;
double **d_Carray;

double **Aarray;
double **Barray;
double **Carray;

hipblasHandle_t handle_dgemm;	

extern "C" void cublasDgemmBatched_wrapper (char transa, char transb, int m, int n,int k, double alpha, const double *A, int lda, int tda, const double *B, int ldb, int tdb, double beta, double *C, int ldc, int tdc, int batchCount)
{


  // printf("CUBLAS m=%d,n=%d,k=%d,batchcount=%d\n",m,n,k,batchCount);
    hipblasStatus_t stat;

 
  hipblasOperation_t op_t1=HIPBLAS_OP_N, op_t2=HIPBLAS_OP_N;

  if (transa=='T' || transa=='t')	
    op_t1=HIPBLAS_OP_T;

  if (transb=='T' || transb=='t')
    op_t2=HIPBLAS_OP_T;


  //double **Aarray = (double**) malloc(batchCount*sizeof(double*));
  //double **Barray = (double**) malloc(batchCount*sizeof(double*));
  //double **Carray = (double**) malloc(batchCount*sizeof(double*));



  if (!alreadyAllocated_dgemm_handle){
     stat = hipblasCreate(&handle_dgemm);
     if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        //return EXIT_FAILURE;
    }
  }
  alreadyAllocated_dgemm_handle=true;

  if (!alreadyAllocated_dgemm){
    hipError_t errcm1 = hipHostMalloc(&Aarray,batchCount*sizeof(double*), hipHostMallocDefault);
    hipError_t errcm2 = hipHostMalloc(&Barray,batchCount*sizeof(double*), hipHostMallocDefault);
    hipError_t errcm3 = hipHostMalloc(&Carray,batchCount*sizeof(double*), hipHostMallocDefault);
        
    hipError_t errcm4 = hipMalloc(&d_Aarray,batchCount*sizeof(double*));
    hipError_t errcm5 = hipMalloc(&d_Barray,batchCount*sizeof(double*));
    hipError_t errcm6 = hipMalloc(&d_Carray,batchCount*sizeof(double*));
   }
  alreadyAllocated_dgemm=true;

  int i;
  for(i=0;i<batchCount;i++){
    Aarray[i]=(double*) &(A[i*lda*tda]);
    Barray[i]=(double*) &(B[i*ldb*tdb]);
    Carray[i]=(double*) &(C[i*ldc*tdc]);
  }

  hipError_t err1 = hipMemcpy(d_Aarray,Aarray,batchCount*sizeof(double*),hipMemcpyHostToDevice);
  hipError_t err2 = hipMemcpy(d_Barray,Barray,batchCount*sizeof(double*),hipMemcpyHostToDevice);
  hipError_t err3 = hipMemcpy(d_Carray,Carray,batchCount*sizeof(double*),hipMemcpyHostToDevice);
  hipDeviceSynchronize();


  hipblasDgemmStridedBatched(handle_dgemm,op_t1,op_t2,m,n,k,&alpha,A,lda, lda*tda,B,ldb,ldb*tdb,&beta,C,ldc,ldc*tdc,batchCount);

  hipDeviceSynchronize();
  
  //cudaFree(Aarray);
  //cudaFree(Barray);
  //cudaFree(Carray);
  
  //cudaFree(d_Aarray);
  //cudaFree(d_Barray);
  //cudaFree(d_Carray);
  //cublasDestroy(handle_dgemm);
  
  
}


bool alreadyAllocated_sgemm=false;
bool alreadyAllocated_sgemm_handle=false;

float **d_Aarray_sgemm;
float **d_Barray_sgemm;
float **d_Carray_sgemm;

float **Aarray_sgemm;
float **Barray_sgemm;
float **Carray_sgemm;

hipblasHandle_t handle_sgemm;	

extern "C" void cublasSgemmBatched_wrapper (char transa, char transb, int m, int n,int k, float alpha, const float *A, int lda, int tda, const float *B, int ldb, int tdb, float beta, float *C, int ldc, int tdc, int batchCount)
{

  // printf("CUBLAS m=%d,n=%d,k=%d,batchcount=%d\n",m,n,k,batchCount);

  hipblasOperation_t op_t1=HIPBLAS_OP_N, op_t2=HIPBLAS_OP_N;

  if (transa=='T' || transa=='t')		
    op_t1=HIPBLAS_OP_T;

  if (transb=='T' || transb=='t')		
    op_t2=HIPBLAS_OP_T;

  //float **Aarray_sgemm = (float**) malloc(batchCount*sizeof(float*));
  //float **Barray_sgemm = (float**) malloc(batchCount*sizeof(float*));
  //float **Carray_sgemm = (float**) malloc(batchCount*sizeof(float*));

  if (!alreadyAllocated_sgemm_handle){
    hipblasCreate(&handle_sgemm);
    alreadyAllocated_sgemm_handle=true;
  }

  if (!alreadyAllocated_sgemm){
    hipHostMalloc(&Aarray_sgemm,batchCount*sizeof(float*), hipHostMallocDefault);
    hipHostMalloc(&Barray_sgemm,batchCount*sizeof(float*), hipHostMallocDefault);
    hipHostMalloc(&Carray_sgemm,batchCount*sizeof(float*), hipHostMallocDefault);
    alreadyAllocated_sgemm=true;
  }

  hipMalloc(&d_Aarray_sgemm,batchCount*sizeof(float*));
  hipMalloc(&d_Barray_sgemm,batchCount*sizeof(float*));
  hipMalloc(&d_Carray_sgemm,batchCount*sizeof(float*));

  int i;
  for(i=0;i<batchCount;i++){
    Aarray_sgemm[i]=(float*) &(A[i*lda*tda]);
    Barray_sgemm[i]=(float*) &(B[i*ldb*tdb]);
    Carray_sgemm[i]=(float*) &(C[i*ldc*tdc]);
  }
  hipMemcpy(d_Aarray_sgemm,Aarray_sgemm,batchCount*sizeof(float*),hipMemcpyHostToDevice);
  hipMemcpy(d_Barray_sgemm,Barray_sgemm,batchCount*sizeof(float*),hipMemcpyHostToDevice);
  hipMemcpy(d_Carray_sgemm,Carray_sgemm,batchCount*sizeof(float*),hipMemcpyHostToDevice);

  hipblasSgemmStridedBatched(handle_sgemm,op_t1,op_t2,m,n,k,&alpha,A,lda, lda*tda,B,ldb,ldb*tdb,&beta,C,ldc,ldc*tdc,batchCount);

  //printf("after sgemm\n");
  hipDeviceSynchronize();
  
  //cudaFree(Aarray_sgemm);
  //cudaFree(Barray_sgemm);
  //cudaFree(Carray_sgemm);
  
  hipFree(d_Aarray_sgemm);
  hipFree(d_Barray_sgemm);
  hipFree(d_Carray_sgemm);
  //cublasDestroy(handle_sgemm);
  
}

