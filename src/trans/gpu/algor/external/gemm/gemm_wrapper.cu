
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <stdio.h>

#define CUDA_CHECK(e)                                                          \
  {                                                                            \
    hipError_t err = (e);                                                     \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA error: %s, line %d, %s: %s\n", __FILE__, __LINE__, \
              #e, hipGetErrorString(err));                                    \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  }
#define CUBLAS_CHECK(e)                                                        \
  {                                                                            \
    hipblasStatus_t err = (e);                                                  \
    if (err != HIPBLAS_STATUS_SUCCESS) {                                        \
      fprintf(stderr, "CUBLAS error: %s, line %d, %s: %i\n", __FILE__,         \
              __LINE__, #e, err);                                              \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  }

extern "C" {
void cublas_dgemm_wrapper(hipblasOperation_t transa, hipblasOperation_t transb,
                          int m, int n, int k, double alpha, const double *A,
                          int lda, int tda, const double *B, int ldb, int tdb,
                          double beta, double *C, int ldc, int tdc,
                          int batchCount) {
  static hipblasHandle_t handle = nullptr;
  if (!handle)
    CUBLAS_CHECK(hipblasCreate(&handle));

  CUBLAS_CHECK(hipblasDgemmStridedBatched(handle, transa, transb, m, n, k,
                                         &alpha, A, lda, tda, B, ldb, tdb,
                                         &beta, C, ldc, tdc, batchCount));
  CUDA_CHECK(hipDeviceSynchronize());
}

void cublas_sgemm_wrapper(hipblasOperation_t transa, hipblasOperation_t transb,
                          int m, int n, int k, float alpha, const float *A,
                          int lda, int tda, const float *B, int ldb, int tdb,
                          float beta, float *C, int ldc, int tdc,
                          int batchCount) {
  static hipblasHandle_t handle = nullptr;
  if (!handle)
    CUBLAS_CHECK(hipblasCreate(&handle));

  CUBLAS_CHECK(hipblasSgemmStridedBatched(handle, transa, transb, m, n, k,
                                         &alpha, A, lda, tda, B, ldb, tdb,
                                         &beta, C, ldc, tdc, batchCount));
  CUDA_CHECK(hipDeviceSynchronize());
}
}
