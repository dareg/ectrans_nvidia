#include <stdio.h>

#include <iostream>
#include <memory>
#include <type_traits>
#include <unordered_map>

#include "hipblas.h"
#include "cutlass/gemm/device/gemm.h"

constexpr bool use_cutlass = true;

#define CUDA_CHECK(e)                                                          \
  {                                                                            \
    hipError_t err = (e);                                                     \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA error: %s, line %d, %s: %s\n", __FILE__, __LINE__, \
              #e, hipGetErrorString(err));                                    \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  }
#define CUBLAS_CHECK(e)                                                \
  {                                                                    \
    hipblasStatus_t err = (e);                                          \
    if (err != HIPBLAS_STATUS_SUCCESS) {                                \
      fprintf(stderr, "CUBLAS error: %s, line %d, %s: %i\n", __FILE__, \
              __LINE__, #e, err);                                      \
      exit(EXIT_FAILURE);                                              \
    }                                                                  \
  }
#define CUTLASS_CHECK(e)                                                \
  {                                                                     \
    cutlass::Status err = (e);                                          \
    if (err != cutlass::Status::kSuccess) {                             \
      fprintf(stderr, "CUTLASS error: %s, line %d, %s: %i\n", __FILE__, \
              __LINE__, #e, (int)err);                                  \
      exit(EXIT_FAILURE);                                               \
    }                                                                   \
  }

namespace {
namespace detail {
struct pair_hash {
  std::size_t operator()(const std::pair<int, int> &p) const {
    return p.first * 10000 + p.second;
  }
};
}  // namespace detail

// this version is using cuda graphs and caches the graphs
template <typename Gemm, typename Real>
void run_group_graph(Gemm &&gemm, int m, int *n, int *k, Real alpha,
                     const Real *A, int lda, int *offsetsA, const Real *B,
                     int ldb, int *offsetsB, Real beta, Real *C, int ldc,
                     int *offsetsC, int batchCount, int blas_id = -1) {
  // we store at most one graph per "m" (# fields) and "blas id"
  static std::unordered_map<std::pair<int, int>, hipGraphExec_t,
                            detail::pair_hash>
      graphCache;

  // we also store A, B, and C and recreate the graph if they change
  static std::unordered_map<
      std::pair<int, int>, std::tuple<Real const *, Real const *, Real const *>,
      detail::pair_hash>
      ptrCache;

  auto key = std::make_pair(m, blas_id);

  auto ptrs = ptrCache.find(key);
  if (ptrs != ptrCache.end() &&
      (std::get<0>(ptrs->second) != A || std::get<1>(ptrs->second) != B ||
       std::get<2>(ptrs->second) != C)) {
    // the plan is cached, but the pointers are not correct. we remove and
    // delete the graph, but we keep the cublas handles, if this happens more
    // often, we should cache this...
    std::cout << "WARNING: POINTER CHANGE --> THIS MIGHT BE SLOW" << std::endl;
    CUDA_CHECK(hipGraphExecDestroy(graphCache[key]));
    graphCache.erase(key);
    ptrCache.erase(key);
  }

  auto graph = graphCache.find(key);
  if (graph == graphCache.end()) {
    // this graph does not exist yet
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    hipGraph_t new_graph;
    hipGraphCreate(&new_graph, 0);
    for (int i = 0; i < batchCount; ++i) {
      if (m == 0 || n[i] == 0 || k[i] == 0) continue;

      CUDA_CHECK(hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal));
      gemm(stream, m, n[i], k[i], alpha, A + offsetsA[i], lda, B + offsetsB[i],
           ldb, beta, C + offsetsC[i], ldc);
      hipGraph_t my_graph;
      CUDA_CHECK(hipStreamEndCapture(stream, &my_graph));
      hipGraphNode_t my_node;
      CUDA_CHECK(hipGraphAddChildGraphNode(&my_node, new_graph, nullptr, 0,
                                            my_graph));
    }
    hipGraphExec_t instance;
    CUDA_CHECK(hipGraphInstantiate(&instance, new_graph, NULL, NULL, 0));
    CUDA_CHECK(hipStreamDestroy(stream));
    CUDA_CHECK(hipGraphDestroy(new_graph));

    graphCache.insert({key, instance});
    ptrCache.insert({key, std::make_tuple(A, B, C)});
  }

  CUDA_CHECK(hipGraphLaunch(graphCache.at(key), 0));
}

// stupid simple gemm calls
template <typename Gemm, typename Real>
void run_group(Gemm &&gemm, int m, int *n, int *k, Real alpha, const Real *A,
               int lda, int *offsetsA, const Real *B, int ldb, int *offsetsB,
               Real beta, Real *C, int ldc, int *offsetsC, int batchCount,
               int = -1) {
  for (int i = 0; i < batchCount; ++i) {
    if (m == 0 || n[i] == 0 || k[i] == 0) continue;
    gemm(0, m, n[i], k[i], alpha, A + offsetsA[i], lda, B + offsetsB[i], ldb,
         beta, C + offsetsC[i], ldc);
  }
}

template <typename CutlassGemm>
CutlassGemm &get_cutlass_handle() {
  static auto handle = std::make_unique<CutlassGemm>();
  return *handle;
}

namespace detail {
template <hipblasOperation_t TransA, hipblasOperation_t TransB>
class cutlass_sgemm_grouped {
#if 0
  // we will enable this later (this ifdefs did not work, so I am going to enable this properly ltaer)
  // this was verified using Ampere and uses 3XTF32
  static constexpr int AlignmentA = 4;
  static constexpr int AlignmentB = 4;
  using ThreadblockShape = cutlass::gemm::GemmShape<128, 64, 32>;
  using WarpShape = cutlass::gemm::GemmShape<64, 32, 32>;
  using InstructionShape = cutlass::gemm::GemmShape<16, 8, 8>;
  using OperatorClass = cutlass::arch::OpClassTensorOp;
  using MyOp = cutlass::arch::OpMultiplyAddFastF32;

  using Gemm = cutlass::gemm::device::Gemm<
      float,
      std::conditional_t<TransA == HIPBLAS_OP_N, cutlass::layout::ColumnMajor,
                         cutlass::layout::RowMajor>,  //
      float,
      std::conditional_t<TransB == HIPBLAS_OP_N, cutlass::layout::ColumnMajor,
                         cutlass::layout::RowMajor>,  //
      float, cutlass::layout::ColumnMajor,            //
      float,                                          //
      OperatorClass, cutlass::arch::Sm80,             //
      ThreadblockShape, WarpShape, InstructionShape,  //
      cutlass::epilogue::thread::LinearCombination<   //
          float,                                      //
          128 / cutlass::sizeof_bits<float>::value,
          float,                                                     //
          float                                                      //
          >,                                                         //
      cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,  //
      3,                                                             //
      AlignmentA,                                                    //
      AlignmentB,                                                    //
      true,                                                          //
      MyOp                                                           //
      >;
  static constexpr int sz_align = 8;
#else
  // this was verified using Volta and uses FP32
  static constexpr int AlignmentA = 1;
  static constexpr int AlignmentB = 1;
  using ThreadblockShape = cutlass::gemm::GemmShape<128, 128, 8>;
  using WarpShape = cutlass::gemm::GemmShape<32, 32, 8>;
  using InstructionShape = cutlass::gemm::GemmShape<1, 1, 1>;
  using OperatorClass = cutlass::arch::OpClassSimt;
  using MyOp = cutlass::arch::OpMultiplyAdd;

  using Gemm = cutlass::gemm::device::Gemm<
      float,  //
      std::conditional_t<TransA == HIPBLAS_OP_N, cutlass::layout::ColumnMajor,
                         cutlass::layout::RowMajor>,  //
      float,                                          //
      std::conditional_t<TransB == HIPBLAS_OP_N, cutlass::layout::ColumnMajor,
                         cutlass::layout::RowMajor>,                 //
      float, cutlass::layout::ColumnMajor,                           //
      float,                                                         //
      OperatorClass, cutlass::arch::Sm70,                            //
      ThreadblockShape, WarpShape, InstructionShape,                 //
      cutlass::epilogue::thread::LinearCombination<                  //
          float,                                                     //
          1,                                                         //
          float,                                                     //
          float                                                      //
          >,                                                         //
      cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,  //
      2,                                                             //
      AlignmentA,                                                    //
      AlignmentB,                                                    //
      true,                                                          //
      MyOp                                                           //
      >;
  static constexpr int sz_align = 1;
#endif

 public:
  void operator()(hipStream_t stream, int m, int n, int k, float alpha,
                  const float *A, int lda, const float *B, int ldb, float beta,
                  float *C, int ldc) const {
    auto &gemm_op = get_cutlass_handle<Gemm>();
    CUTLASS_CHECK(gemm_op(
        {//
         {(m + sz_align - 1) / sz_align * sz_align,
          (n + sz_align - 1) / sz_align * sz_align,
          (k + sz_align - 1) / sz_align * sz_align},
         {const_cast<float *>(A), lda},
         {const_cast<float *>(B), ldb},
         {C, ldc},
         {C, ldc},
         {alpha, beta}},
        nullptr, stream));
  }
};

}  // namespace detail
template <hipblasOperation_t TransA, hipblasOperation_t TransB>
void cutlass_sgemm_wrapper_grouped_op(int blas_id, int m, int *n, int *k,
                                      float alpha, const float *A, int lda,
                                      int *offsetsA, const float *B, int ldb,
                                      int *offsetsB, float beta, float *C,
                                      int ldc, int *offsetsC, int batchCount) {
  using namespace detail;
  run_group_graph(cutlass_sgemm_grouped<TransA, TransB>(), m, n, k, alpha, A,
                  lda, offsetsA, B, ldb, offsetsB, beta, C, ldc, offsetsC,
                  batchCount, blas_id);
}
void cutlass_sgemm_wrapper_grouped(int blas_id, hipblasOperation_t transa,
                                   hipblasOperation_t transb, int m, int *n,
                                   int *k, float alpha, const float *A, int lda,
                                   int *offsetsA, const float *B, int ldb,
                                   int *offsetsB, float beta, float *C, int ldc,
                                   int *offsetsC, int batchCount) {
  if (transa == HIPBLAS_OP_N && transb == HIPBLAS_OP_N)
    cutlass_sgemm_wrapper_grouped_op<HIPBLAS_OP_N, HIPBLAS_OP_N>(
        blas_id, m, n, k, alpha, A, lda, offsetsA, B, ldb, offsetsB, beta, C,
        ldc, offsetsC, batchCount);
  else if (transa == HIPBLAS_OP_N && transb == HIPBLAS_OP_T)
    cutlass_sgemm_wrapper_grouped_op<HIPBLAS_OP_N, HIPBLAS_OP_T>(
        blas_id, m, n, k, alpha, A, lda, offsetsA, B, ldb, offsetsB, beta, C,
        ldc, offsetsC, batchCount);
  else if (transa == HIPBLAS_OP_T && transb == HIPBLAS_OP_N)
    cutlass_sgemm_wrapper_grouped_op<HIPBLAS_OP_T, HIPBLAS_OP_N>(
        blas_id, m, n, k, alpha, A, lda, offsetsA, B, ldb, offsetsB, beta, C,
        ldc, offsetsC, batchCount);
  else if (transa == HIPBLAS_OP_T && transb == HIPBLAS_OP_T)
    cutlass_sgemm_wrapper_grouped_op<HIPBLAS_OP_T, HIPBLAS_OP_T>(
        blas_id, m, n, k, alpha, A, lda, offsetsA, B, ldb, offsetsB, beta, C,
        ldc, offsetsC, batchCount);
  else
    assert(false);
}

namespace detail {
hipblasHandle_t get_cublas_handle() {
  static hipblasHandle_t handle;
  if (!handle) CUBLAS_CHECK(hipblasCreate(&handle));
  return handle;
}
template <typename Real>
struct cublas_gemm_grouped {
 public:
  cublas_gemm_grouped(hipblasOperation_t transa, hipblasOperation_t transb)
      : transa_(transa), transb_(transb) {
    // we need to get the cublas handle here, otherwise this could be created
    // during graph capturing
    get_cublas_handle();
  };
  void operator()(hipStream_t stream, int m, int n, int k, Real alpha,
                  const Real *A, int lda, const Real *B, int ldb, Real beta,
                  Real *C, int ldc) const {
    hipblasHandle_t handle = get_cublas_handle();
    CUBLAS_CHECK(hipblasSetStream(handle, stream));

    if constexpr (std::is_same<Real, float>::value)
      CUBLAS_CHECK(hipblasSgemm(handle, transa_, transb_, m, n, k, &alpha, A,
                               lda, B, ldb, &beta, C, ldc));
    if constexpr (std::is_same<Real, double>::value)
      CUBLAS_CHECK(hipblasDgemm(handle, transa_, transb_, m, n, k, &alpha, A,
                               lda, B, ldb, &beta, C, ldc));
  }

 private:
  hipblasOperation_t transa_, transb_;
};
}  // namespace detail
void cublas_sgemm_wrapper_grouped(int blas_id, hipblasOperation_t transa,
                                  hipblasOperation_t transb, int m, int *n,
                                  int *k, float alpha, const float *A, int lda,
                                  int *offsetsA, const float *B, int ldb,
                                  int *offsetsB, float beta, float *C, int ldc,
                                  int *offsetsC, int batchCount) {
  using namespace detail;
  run_group_graph(cublas_gemm_grouped<float>(transa, transb), m, n, k, alpha, A,
                  lda, offsetsA, B, ldb, offsetsB, beta, C, ldc, offsetsC,
                  batchCount, blas_id);
}
void cublas_dgemm_wrapper_grouped(int blas_id, hipblasOperation_t transa,
                                  hipblasOperation_t transb, int m, int *n,
                                  int *k, double alpha, const double *A,
                                  int lda, int *offsetsA, const double *B,
                                  int ldb, int *offsetsB, double beta,
                                  double *C, int ldc, int *offsetsC,
                                  int batchCount) {
  using namespace detail;
  run_group_graph(cublas_gemm_grouped<double>(transa, transb), m, n, k, alpha,
                  A, lda, offsetsA, B, ldb, offsetsB, beta, C, ldc, offsetsC,
                  batchCount, blas_id);
}
}  // namespace

extern "C" {
void cublas_dgemm_wrapper(hipblasOperation_t transa, hipblasOperation_t transb,
                          int m, int n, int k, double alpha, const double *A,
                          int lda, int tda, const double *B, int ldb, int tdb,
                          double beta, double *C, int ldc, int tdc,
                          int batchCount) {
  static hipblasHandle_t handle = nullptr;
  if (!handle) CUBLAS_CHECK(hipblasCreate(&handle));

  CUBLAS_CHECK(hipblasDgemmStridedBatched(handle, transa, transb, m, n, k,
                                         &alpha, A, lda, tda, B, ldb, tdb,
                                         &beta, C, ldc, tdc, batchCount));
}

void cublas_sgemm_wrapper(hipblasOperation_t transa, hipblasOperation_t transb,
                          int m, int n, int k, float alpha, const float *A,
                          int lda, int tda, const float *B, int ldb, int tdb,
                          float beta, float *C, int ldc, int tdc,
                          int batchCount) {
  static hipblasHandle_t handle = nullptr;
  if (!handle) CUBLAS_CHECK(hipblasCreate(&handle));

  CUBLAS_CHECK(hipblasSgemmStridedBatched(handle, transa, transb, m, n, k,
                                         &alpha, A, lda, tda, B, ldb, tdb,
                                         &beta, C, ldc, tdc, batchCount));
}

void blas_sgemm_wrapper_grouped(int blas_id, hipblasOperation_t transa,
                                hipblasOperation_t transb, int m, int *n, int *k,
                                float alpha, const float *A, int lda,
                                int *offsetsA, const float *B, int ldb,
                                int *offsetsB, float beta, float *C, int ldc,
                                int *offsetsC, int batchCount) {
  if (use_cutlass)
    cutlass_sgemm_wrapper_grouped(blas_id, transa, transb, m, n, k, alpha, A,
                                  lda, offsetsA, B, ldb, offsetsB, beta, C, ldc,
                                  offsetsC, batchCount);
  else
    cublas_sgemm_wrapper_grouped(blas_id, transa, transb, m, n, k, alpha, A, lda,
                                 offsetsA, B, ldb, offsetsB, beta, C, ldc,
                                 offsetsC, batchCount);
}
void blas_dgemm_wrapper_grouped(int blas_id, hipblasOperation_t transa,
                                hipblasOperation_t transb, int m, int *n, int *k,
                                double alpha, const double *A, int lda,
                                int *offsetsA, const double *B, int ldb,
                                int *offsetsB, double beta, double *C, int ldc,
                                int *offsetsC, int batchCount) {
  cublas_dgemm_wrapper_grouped(blas_id, transa, transb, m, n, k, alpha, A, lda, offsetsA,
                               B, ldb, offsetsB, beta, C, ldc, offsetsC,
                               batchCount);
}
}
