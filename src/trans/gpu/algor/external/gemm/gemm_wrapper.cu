
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <stdio.h>

#define CUDA_CHECK(e)                                                          \
  {                                                                            \
    hipError_t err = (e);                                                     \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA error: %s, line %d, %s: %s\n", __FILE__, __LINE__, \
              #e, hipGetErrorString(err));                                    \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  }
#define CUBLAS_CHECK(e)                                                        \
  {                                                                            \
    hipblasStatus_t err = (e);                                                  \
    if (err != HIPBLAS_STATUS_SUCCESS) {                                        \
      fprintf(stderr, "CUBLAS error: %s, line %d, %s: %i\n", __FILE__,         \
              __LINE__, #e, err);                                              \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  }

extern "C" {
void cublas_dgemm_wrapper(hipblasOperation_t transa, hipblasOperation_t transb,
                          int m, int n, int k, double alpha, const double *A,
                          int lda, int tda, const double *B, int ldb, int tdb,
                          double beta, double *C, int ldc, int tdc,
                          int batchCount) {
  static hipblasHandle_t handle = nullptr;
  if (!handle)
    CUBLAS_CHECK(hipblasCreate(&handle));

  CUBLAS_CHECK(hipblasDgemmStridedBatched(handle, transa, transb, m, n, k,
                                         &alpha, A, lda, tda, B, ldb, tdb,
                                         &beta, C, ldc, tdc, batchCount));
}

void cublas_sgemm_wrapper(hipblasOperation_t transa, hipblasOperation_t transb,
                          int m, int n, int k, float alpha, const float *A,
                          int lda, int tda, const float *B, int ldb, int tdb,
                          float beta, float *C, int ldc, int tdc,
                          int batchCount) {
  static hipblasHandle_t handle = nullptr;
  if (!handle)
    CUBLAS_CHECK(hipblasCreate(&handle));

  CUBLAS_CHECK(hipblasSgemmStridedBatched(handle, transa, transb, m, n, k,
                                         &alpha, A, lda, tda, B, ldb, tdb,
                                         &beta, C, ldc, tdc, batchCount));
}

void cublas_sgemm_wrapper_grouped(hipblasOperation_t transa,
                                  hipblasOperation_t transb, int m, int *n,
                                  int *k, float alpha, const float *A, int lda,
                                  int tda, const float *B, int ldb, int tdb,
                                  float beta, float *C, int ldc, int tdc,
                                  int batchCount) {
  static hipblasHandle_t handle = nullptr;
  if (!handle)
    CUBLAS_CHECK(hipblasCreate(&handle));

  for (int i = 0; i < batchCount; ++i) {
    CUBLAS_CHECK(hipblasSgemm(handle, transa, transb, m, n[i], k[i], &alpha,
                             A + i * tda, lda, B + i * tdb, ldb, &beta,
                             C + i * tdc, ldc));
  }
}
void cublas_dgemm_wrapper_grouped(hipblasOperation_t transa,
                                  hipblasOperation_t transb, int m, int *n,
                                  int *k, double alpha, const double *A,
                                  int lda, int tda, const double *B, int ldb,
                                  int tdb, double beta, double *C, int ldc,
                                  int tdc, int batchCount) {
  static hipblasHandle_t handle = nullptr;
  if (!handle)
    CUBLAS_CHECK(hipblasCreate(&handle));

  for (int i = 0; i < batchCount; ++i) {
    CUBLAS_CHECK(hipblasDgemm(handle, transa, transb, m, n[i], k[i], &alpha,
                             A + i * tda, lda, B + i * tdb, ldb, &beta,
                             C + i * tdc, ldc));
  }
}
}
