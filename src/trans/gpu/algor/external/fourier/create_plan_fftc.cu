
#include <hip/hip_runtime.h>
#include "hipfft/hipfft.h"
#include "stdio.h"
static const char *_cudaGetErrorEnum(hipfftResult error) {
  switch (error) {
  case HIPFFT_SUCCESS:
    return "CUFFT_SUCCESS";

  case HIPFFT_INVALID_PLAN:
    return "CUFFT_INVALID_PLAN";

  case HIPFFT_ALLOC_FAILED:
    return "CUFFT_ALLOC_FAILED";

  case HIPFFT_INVALID_TYPE:
    return "CUFFT_INVALID_TYPE";

  case HIPFFT_INVALID_VALUE:
    return "CUFFT_INVALID_VALUE";

  case HIPFFT_INTERNAL_ERROR:
    return "CUFFT_INTERNAL_ERROR";

  case HIPFFT_EXEC_FAILED:
    return "CUFFT_EXEC_FAILED";

  case HIPFFT_SETUP_FAILED:
    return "CUFFT_SETUP_FAILED";

  case HIPFFT_INVALID_SIZE:
    return "CUFFT_INVALID_SIZE";

  case HIPFFT_UNALIGNED_DATA:
    return "CUFFT_UNALIGNED_DATA";
  }

  return "<unknown>";
}
#define CUFFT_CHECK(e) { \
	hipfftResult_t err = (e); \
	if (err != HIPFFT_SUCCESS) \
	{ \
		fprintf(stderr, "CUFFT error: %s, line %d, %s: %s\n", \
			__FILE__, __LINE__, #e, _cudaGetErrorEnum(err)); \
		exit(EXIT_FAILURE); \
	} \
}

void *planWorkspace = nullptr;
static int currentWorkspaceSize = 0;

extern "C" void create_plan_fftc_(hipfftHandle *PLANp, int *ISIGNp, int *Np,
                                  int *LOTp, int *stridep, int *plan_size) {
  int ISIGN = *ISIGNp;
  int N = *Np;
  int LOT = *LOTp;
  int stride = *stridep;

  hipfftHandle plan;

  if (hipDeviceSynchronize() != hipSuccess) {
    fprintf(stderr, "Cuda error: Failed to synchronize\n");
    return;
  }

  int embed[1];
  int dist;

#ifdef TRANS_SINGLE
  cufftType cufft_1 = CUFFT_R2C;
  cufftType cufft_2 = CUFFT_C2R;
#else
  hipfftType cufft_1 = HIPFFT_D2Z;
  hipfftType cufft_2 = HIPFFT_Z2D;
#endif

  embed[0] = 1;
  dist = 1;

  CUFFT_CHECK(hipfftCreate(&plan));

  // Disable auto allocation
  CUFFT_CHECK(hipfftSetAutoAllocation(plan, false));

  // printf("CreatePlan cuFFT\n","N=",N);
  // printf("%s %d \n","plan=",plan);
  // printf("%s %d \n","LOT=",LOT);
  // printf("%s %d \n","ISIGN=",ISIGN);
  // printf("%s %d \n","Np=",*Np);

  if (ISIGN == -1) {
    CUFFT_CHECK(hipfftPlanMany(&plan, 1, &N, embed, stride, dist, embed,
                                stride, dist, cufft_1, LOT));
  } else if (ISIGN == 1) {
    CUFFT_CHECK(hipfftPlanMany(&plan, 1, &N, embed, stride, dist, embed,
                                stride, dist, cufft_2, LOT));
  } else {
    abort();
  }

  // get size used by this plan
  size_t thisWorkplanSize;
  CUFFT_CHECK(hipfftGetSize(plan, &thisWorkplanSize));

  // check if this the work space is sufficiently large
  if (thisWorkplanSize > currentWorkspaceSize) {
    hipDeviceSynchronize();
    hipFree(planWorkspace);
    hipMalloc(&planWorkspace, thisWorkplanSize);
    currentWorkspaceSize = thisWorkplanSize;
  }

  if (hipDeviceSynchronize() != hipSuccess) {
    fprintf(stderr, "Cuda error: Failed to synchronize\n");
    return;
  }

  *PLANp = plan;
  *plan_size = thisWorkplanSize;

  return;
}
