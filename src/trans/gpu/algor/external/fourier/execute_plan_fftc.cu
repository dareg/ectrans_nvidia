
#include <hip/hip_runtime.h>
#include "hipfft/hipfft.h"
#include "stdio.h"
static const char *_cudaGetErrorEnum(hipfftResult error) {
  switch (error) {
  case HIPFFT_SUCCESS:
    return "CUFFT_SUCCESS";

  case HIPFFT_INVALID_PLAN:
    return "CUFFT_INVALID_PLAN";

  case HIPFFT_ALLOC_FAILED:
    return "CUFFT_ALLOC_FAILED";

  case HIPFFT_INVALID_TYPE:
    return "CUFFT_INVALID_TYPE";

  case HIPFFT_INVALID_VALUE:
    return "CUFFT_INVALID_VALUE";

  case HIPFFT_INTERNAL_ERROR:
    return "CUFFT_INTERNAL_ERROR";

  case HIPFFT_EXEC_FAILED:
    return "CUFFT_EXEC_FAILED";

  case HIPFFT_SETUP_FAILED:
    return "CUFFT_SETUP_FAILED";

  case HIPFFT_INVALID_SIZE:
    return "CUFFT_INVALID_SIZE";

  case HIPFFT_UNALIGNED_DATA:
    return "CUFFT_UNALIGNED_DATA";
  }

  return "<unknown>";
}

#define CUFFT_CHECK(e) { \
	hipfftResult_t err = (e); \
	if (err != HIPFFT_SUCCESS) \
	{ \
		fprintf(stderr, "CUFFT error: %s, line %d, %s: %s\n", \
			__FILE__, __LINE__, #e, _cudaGetErrorEnum(err)); \
		exit(EXIT_FAILURE); \
	} \
}

extern void *planWorkspace;

extern "C" void
#ifdef TRANS_SINGLE
execute_plan_fftc_(cufftHandle *PLANp, int *ISIGNp, cufftComplex *data_in,
                   cufftComplex *data_out)
#else
execute_plan_fftc_(hipfftHandle *PLANp, int *ISIGNp, hipfftDoubleComplex *data_in,
                   hipfftDoubleComplex *data_out)
#endif
{
  hipfftHandle plan = *PLANp;
  int ISIGN = *ISIGNp;

  CUFFT_CHECK(hipfftSetWorkArea(plan, planWorkspace));

  if (ISIGN == -1) {
#ifdef TRANS_SINGLE
    CUFFT_CHECK(cufftExecR2C(plan, (cufftReal *)data_in, data_out));
#else
    CUFFT_CHECK(hipfftExecD2Z(plan, (hipfftDoubleReal *)data_in, data_out));
#endif
  } else if (ISIGN == 1) {
#ifdef TRANS_SINGLE
    CUFFT_CHECK(cufftExecC2R(plan, data_in, (cufftReal *)data_out));
#else
    CUFFT_CHECK(hipfftExecZ2D(plan, data_in, (hipfftDoubleReal *)data_out));
#endif
  } else {
    abort();
  }
}
