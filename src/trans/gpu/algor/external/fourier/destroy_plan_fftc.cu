#define cufftSafeCall(err) __cufftSafeCall(err, __FILE__, __LINE__)

#include <hip/hip_runtime.h>
#include "hipfft/hipfft.h"
#include "stdio.h"
static const char *_cudaGetErrorEnum(hipfftResult error) {
  switch (error) {
  case HIPFFT_SUCCESS:
    return "CUFFT_SUCCESS";

  case HIPFFT_INVALID_PLAN:
    return "CUFFT_INVALID_PLAN";

  case HIPFFT_ALLOC_FAILED:
    return "CUFFT_ALLOC_FAILED";

  case HIPFFT_INVALID_TYPE:
    return "CUFFT_INVALID_TYPE";

  case HIPFFT_INVALID_VALUE:
    return "CUFFT_INVALID_VALUE";

  case HIPFFT_INTERNAL_ERROR:
    return "CUFFT_INTERNAL_ERROR";

  case HIPFFT_EXEC_FAILED:
    return "CUFFT_EXEC_FAILED";

  case HIPFFT_SETUP_FAILED:
    return "CUFFT_SETUP_FAILED";

  case HIPFFT_INVALID_SIZE:
    return "CUFFT_INVALID_SIZE";

  case HIPFFT_UNALIGNED_DATA:
    return "CUFFT_UNALIGNED_DATA";
  }

  return "<unknown>";
}

inline void __cufftSafeCall(hipfftResult err, const char *file, const int line) {
  if (HIPFFT_SUCCESS != err) {
    fprintf(stderr, "CUFFT error at 1\n");
    fprintf(stderr, "CUFFT error in file '%s'\n", __FILE__);
    fprintf(stderr, "CUFFT error at 2\n");
    /*fprintf(stderr, "CUFFT error line '%s'\n",__LINE__);*/
    fprintf(stderr, "CUFFT error at 3\n");
    /*fprintf(stderr, "CUFFT error in file '%s', line %d\n %s\nerror %d:
    %s\nterminating!\n",__FILE__, __LINE__,err, \
    _cudaGetErrorEnum(err)); \*/
    fprintf(stderr, "CUFFT error %d: %s\nterminating!\n", err,
            _cudaGetErrorEnum(err));
    hipDeviceReset();
    return;
  }
}

extern "C" void destroy_plan_fftc_(hipfftHandle *PLANp) {
  hipfftHandle plan = *PLANp;

  if (hipDeviceSynchronize() != hipSuccess) {
    fprintf(stderr, "Cuda error: Failed to synchronize\n");
    return;
  }

  cufftSafeCall(hipfftDestroy(plan));

  if (hipDeviceSynchronize() != hipSuccess) {
    fprintf(stderr, "Cuda error: Failed to synchronize\n");
    return;
  }
}
